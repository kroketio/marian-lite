#include "hip/hip_runtime.h"
/* All or part of this file was contributed by Intel under license:
 *   Copyright (C) 2017-2018 Intel Corporation
 *   SPDX-License-Identifier: MIT
 */

#include <iostream>

#include "marian-lite/translator/nth_element.h"

#include <hip/hip_runtime.h>
#include "marian-lite/tensors/gpu/cuda_helpers.h"

namespace marian {

#define UNROLL_MAXARG_LOOP(n, max)       \
  if(tid < (n) && tid + (n) < (max)) {   \
    if(sdata[tid + (n)] > sdata[tid]) {  \
      sdata[tid] = sdata[tid + (n)];     \
      indices[tid] = indices[tid + (n)]; \
    }                                    \
  }

template <typename T>
__global__ void gMaxElement(float* d_out,
                            int* d_ind,
                            T* d_in, // this is the probs array, only one with type float or half
                            int numBatches,
                            int* batchFirstElementIdxs,
                            float disabledPathScore) // disabledPathScore is used to blank out found values, type-dependent
{
  extern __shared__ float sdata[];
  __shared__ int indices[512];

  int tid = threadIdx.x;

  for(int batchIdx = 0; batchIdx < numBatches; ++batchIdx) {
    int begin = batchFirstElementIdxs[batchIdx];
    int end = batchFirstElementIdxs[batchIdx + 1];

    int i = begin + blockIdx.x * (blockDim.x * 2) + tid;

    sdata[tid] = disabledPathScore;

    if(i < end) {
      sdata[tid] = (float)d_in[i];
      indices[tid] = i;
    }

    if(i + blockDim.x < end) {
      float a = (float)d_in[i];
      float b = (float)d_in[i + blockDim.x];
      if(a > b) {
        sdata[tid] = a;
        indices[tid] = i;
      } else {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while(i + 2 * gridDim.x * blockDim.x < end) {
      i += 2 * gridDim.x * blockDim.x;

      float a = (float)d_in[i];
      if(a > sdata[tid]) {
        sdata[tid] = a;
        indices[tid] = i;
      }

      if(i + blockDim.x < end) {
        float b = (float)d_in[i + blockDim.x];
        if(b > sdata[tid]) {
          sdata[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for(int s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if(tid < s && tid + s < end) {
        if(sdata[tid + s] > sdata[tid]) {
          sdata[tid] = sdata[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP(32, end);
    UNROLL_MAXARG_LOOP(16, end);
    UNROLL_MAXARG_LOOP(8, end);
    UNROLL_MAXARG_LOOP(4, end);
    UNROLL_MAXARG_LOOP(2, end);
    UNROLL_MAXARG_LOOP(1, end);

    if(tid == 0) {
      d_out[blockIdx.x + batchIdx * gridDim.x] = sdata[0];
      d_ind[blockIdx.x + batchIdx * gridDim.x] = indices[0];
    }
    __syncthreads();
  }
}

template <typename T>
__global__ void gMaxElementUpdate(float* binCosts,
                                  int* binIdxs,
                                  T* probs, // should work well enough with half, uses float everywhere else
                                  int* batchFirstElements,
                                  float* outCosts,
                                  int* outIdxs,
                                  int* cumulativeBeamSizes,
                                  int NUM_BLOCKS,
                                  float disabledPathScore) {
  extern __shared__ float sdata[];
  __shared__ int indices[512];
  __shared__ float bestBinCost;
  __shared__ int bestBinCostIdx;

  const int tid = threadIdx.x;
  const int batchIdx = blockIdx.x;
  const int N = batchFirstElements[batchIdx + 1] - batchFirstElements[batchIdx];
  int num_bins = int(N / (2 * 512)) + int(N % (2 * 512) != 0);
  if(num_bins > 500) {
    num_bins = 500;
  }

  for(int pos = cumulativeBeamSizes[batchIdx];
      pos < cumulativeBeamSizes[batchIdx + 1];
      ++pos) {
    int i = tid;

    sdata[tid] = disabledPathScore;

    if(i < num_bins) {
      sdata[tid] = binCosts[batchIdx * NUM_BLOCKS + i];
      indices[tid] = i;
    }

    if(i + blockDim.x < num_bins) {
      float a = binCosts[batchIdx * NUM_BLOCKS + i];
      float b = binCosts[batchIdx * NUM_BLOCKS + i + blockDim.x];
      if(a > b) {
        sdata[tid] = a;
        indices[tid] = i;
      } else {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while(i + 2 * blockDim.x < num_bins) {
      i += 2 * blockDim.x;

      float a = binCosts[batchIdx * NUM_BLOCKS + i];
      if(a > sdata[tid]) {
        sdata[tid] = a;
        indices[tid] = i;
      }

      if(i + blockDim.x < num_bins) {
        float b = binCosts[batchIdx * NUM_BLOCKS + i + blockDim.x];
        if(b > sdata[tid]) {
          sdata[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for(int s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if(tid < s && tid + s < num_bins) {
        if(sdata[tid + s] > sdata[tid]) {
          sdata[tid] = sdata[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP(32, num_bins);
    UNROLL_MAXARG_LOOP(16, num_bins);
    UNROLL_MAXARG_LOOP(8, num_bins);
    UNROLL_MAXARG_LOOP(4, num_bins);
    UNROLL_MAXARG_LOOP(2, num_bins);
    UNROLL_MAXARG_LOOP(1, num_bins);

    if(tid == 0) {
      bestBinCost = sdata[0];
      bestBinCostIdx = batchIdx * NUM_BLOCKS + indices[0];

      probs[binIdxs[bestBinCostIdx]] = disabledPathScore;

      outIdxs[pos] = binIdxs[bestBinCostIdx];
      outCosts[pos] = bestBinCost;
    }

    __syncthreads();

    i = batchFirstElements[batchIdx]
        + (bestBinCostIdx - batchIdx * NUM_BLOCKS) * (blockDim.x * 2) + tid;
    const int dist = num_bins * 2 * blockDim.x;

    sdata[tid] = disabledPathScore;

    if(i < batchFirstElements[batchIdx + 1]) {
      sdata[tid] = (float)probs[i];
      indices[tid] = i;
    }

    if(i + blockDim.x < batchFirstElements[batchIdx + 1]) {
      float a = (float)probs[i];
      float b = (float)probs[i + blockDim.x];
      if(a > b) {
        sdata[tid] = a;
        indices[tid] = i;
      } else {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while(i + dist < batchFirstElements[batchIdx + 1]) {
      i += dist;

      float a = (float)probs[i];
      if(a > sdata[tid]) {
        sdata[tid] = a;
        indices[tid] = i;
      }

      if(i + blockDim.x < batchFirstElements[batchIdx + 1]) {
        float b = (float)probs[i + blockDim.x];
        if(b > sdata[tid]) {
          sdata[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for(int s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if(tid < s && tid + s < batchFirstElements[batchIdx + 1]) {
        if(sdata[tid + s] > sdata[tid]) {
          sdata[tid] = sdata[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP(32, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(16, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(8, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(4, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(2, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(1, batchFirstElements[batchIdx + 1]);

    if(tid == 0) {
      binCosts[bestBinCostIdx] = sdata[0];
      binIdxs[bestBinCostIdx] = indices[0];
    }
    __syncthreads();
  }
}

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < n) {
    int index = indeces[tid];
    d_out[tid] = d_in[index];
  }
}

class NthElementGPU {
public:
  NthElementGPU() = delete;
  NthElementGPU(const NthElementGPU& copy) = delete;

  NthElementGPU(size_t maxBeamSize,
                size_t maxBatchSize,
                DeviceId deviceId)
      : deviceId_(deviceId),
        maxBeamSize_(maxBeamSize), maxBatchSize_(maxBatchSize),
        NUM_BLOCKS(std::min(
            500,
            int(maxBeamSize* MAX_VOCAB_SIZE / (2 * BLOCK_SIZE))
                + int(maxBeamSize* MAX_VOCAB_SIZE % (2 * BLOCK_SIZE) != 0))) {
    // std::cerr << "NthElement::NthElement" << std::endl;

    hipSetDevice(deviceId_.no);

    CUDA_CHECK(hipMalloc((void**)&d_ind, maxBatchSize * NUM_BLOCKS * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_out, maxBatchSize * NUM_BLOCKS * sizeof(float)));

    CUDA_CHECK(hipMalloc((void**)&d_res_idx, maxBatchSize * maxBeamSize * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_res,     maxBatchSize * maxBeamSize * sizeof(float)));

    CUDA_CHECK(hipHostAlloc((void**)&h_res,     maxBeamSize * maxBatchSize * sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc((void**)&h_res_idx, maxBeamSize * maxBatchSize * sizeof(int), hipHostMallocDefault));

    CUDA_CHECK(hipMalloc((void**)&d_breakdown, maxBeamSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_batchPosition, (maxBatchSize + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_cumBeamSizes,  (maxBatchSize + 1) * sizeof(int)));
  }

  ~NthElementGPU() {
    // No CUDA error checking as this is a destructor and we cannot do anything about errors anyway.
    hipSetDevice(deviceId_.no);
    hipFree(d_cumBeamSizes);
    hipFree(d_batchPosition);
    hipFree(d_breakdown);
    hipHostFree(h_res_idx);
    hipHostFree(h_res);
    hipFree(d_res);
    hipFree(d_res_idx);
    hipFree(d_out);
    hipFree(d_ind);
  }

private:
  template <typename T>
  void selectNBest(T* probs,
                   const std::vector<int>& batchFirstElementIdxs,
                   const std::vector<int>& cumulativeBeamSizes,
                   float disabledPathScore) {

    hipSetDevice(deviceId_.no);
    CUDA_CHECK(hipMemcpyAsync(d_batchPosition,
                               batchFirstElementIdxs.data(),
                               batchFirstElementIdxs.size() * sizeof(int),
                               hipMemcpyHostToDevice,
                               /* stream_ */ 0));
    CUDA_CHECK(hipMemcpyAsync(d_cumBeamSizes,
                               cumulativeBeamSizes.data(),
                               cumulativeBeamSizes.size() * sizeof(int),
                               hipMemcpyHostToDevice,
                               /* stream_ */ 0));

    const int numBatches = batchFirstElementIdxs.size() - 1;

    gMaxElement<<<NUM_BLOCKS,
                  BLOCK_SIZE,
                  BLOCK_SIZE * sizeof(float), // shared memory size
                  /* stream_ */ 0>>>(
        d_out, d_ind, probs, numBatches, d_batchPosition, disabledPathScore);

    gMaxElementUpdate<<<numBatches,
                        BLOCK_SIZE,
                        BLOCK_SIZE * sizeof(float),  // shared memory size
                        /* stream_ */ 0>>>(d_out,
                                           d_ind,
                                           probs,
                                           d_batchPosition,
                                           d_res,
                                           d_res_idx,
                                           d_cumBeamSizes,
                                           NUM_BLOCKS,
                                           disabledPathScore);
  }

public:
  void getNBestList(Tensor scores,
                    size_t N,
                    std::vector<float>& outCosts,
                    std::vector<unsigned>& outKeys,
                    const bool isFirst) {
    hipSetDevice(deviceId_.no);

    const auto vocabSize = scores->shape()[-1];
    const auto inputN    = scores->shape()[-2];
    const auto dimBatch  = scores->shape()[-4];
    ABORT_IF(inputN != (isFirst ? 1 : N), "Input tensor has wrong beam dim??"); // @TODO: Remove isFirst argument altogether
    ABORT_IF(vocabSize > MAX_VOCAB_SIZE, "GetNBestList(): actual vocab size {} exceeds MAX_VOCAB_SIZE of {}", vocabSize, MAX_VOCAB_SIZE);
    ABORT_IF(dimBatch > maxBatchSize_, "GetNBestList(): actual batch size {} exceeds initialization parameter {}", dimBatch, maxBatchSize_);
    ABORT_IF(std::max(N, (size_t)inputN) > maxBeamSize_, "GetNBestList(): actual beam size {} exceeds initialization parameter {}", N, maxBeamSize_);

    const std::vector<size_t> beamSizes(dimBatch, N);
    std::vector<int> cumulativeBeamSizes(beamSizes.size() + 1, 0);    
    std::vector<int> batchFirstElementIdxs(beamSizes.size() + 1, 0);

     for(size_t batchIdx = 0; batchIdx < beamSizes.size(); ++batchIdx) {
#if 1
      cumulativeBeamSizes[batchIdx + 1] = (batchIdx + 1) * (int)N;
      batchFirstElementIdxs[batchIdx + 1] += (batchIdx + 1) * inputN * vocabSize;
      ABORT_IF(cumulativeBeamSizes[batchIdx + 1] != cumulativeBeamSizes[batchIdx] + (int)N, "cumulativeBeamSizes wrong??");
      ABORT_IF((isFirst ? batchIdx + 1 : cumulativeBeamSizes[batchIdx + 1]) != (batchIdx + 1) * inputN, "inputN wrong??");
#else
      cumulativeBeamSizes[batchIdx + 1] = cumulativeBeamSizes[batchIdx] + beamSizes[batchIdx];
      ABORT_IF(cumulativeBeamSizes[batchIdx + 1] != (batchIdx + 1) * N, "cumulativeBeamSizes wrong??");
      batchFirstElementIdxs[batchIdx + 1]
          += ((isFirst) ? (batchIdx + 1) : cumulativeBeamSizes[batchIdx + 1]) * vocabSize;
      ABORT_IF((isFirst ? batchIdx + 1 : cumulativeBeamSizes[batchIdx + 1]) != (batchIdx + 1) * inputN, "inputN wrong??");
#endif
    }

    if(scores->type() == Type::float32) {
      float disabledPathScore = NumericLimits<float>(scores->type()).lowest;
      selectNBest(scores->data<float>(), batchFirstElementIdxs, cumulativeBeamSizes, disabledPathScore);
#if COMPILE_FP16
    } else if(scores->type() == Type::float16) {
      float disabledPathScore = NumericLimits<float>(scores->type()).lowest;
      selectNBest(scores->data<half>(), batchFirstElementIdxs, cumulativeBeamSizes, disabledPathScore);
#endif
    } else {
      ABORT("getNBestList not implemented for type {}", scores->type());
    }
    getPairs(dimBatch * N, outKeys, outCosts);
    ABORT_IF(cumulativeBeamSizes.back() != dimBatch * N, "cumulativeBeamSizes.back() wrong??");
  }

private:
  void getPairs(size_t number,
                std::vector<unsigned>& outKeys,
                std::vector<float>& outValues) {
    hipSetDevice(deviceId_.no);
    CUDA_CHECK(hipMemcpyAsync(h_res,
                               d_res,
                               number * sizeof(float),
                               hipMemcpyDeviceToHost,
                               /* stream_ */ 0));
    CUDA_CHECK(hipMemcpyAsync(h_res_idx,
                               d_res_idx,
                               number * sizeof(int),
                               hipMemcpyDeviceToHost,
                               /* stream_ */ 0));
    hipStreamSynchronize(/* stream_ */ 0);

    for(size_t i = 0; i < number; ++i) {
      outKeys.push_back(h_res_idx[i]);
      outValues.push_back(h_res[i]);
    }

    //lastN = number;
  }

  DeviceId deviceId_;

  const int MAX_VOCAB_SIZE = 500000;
  size_t maxBeamSize_;
  size_t maxBatchSize_;

  const int BLOCK_SIZE = 512;
  const int NUM_BLOCKS;

  int* d_ind;           // [maxBatchSize * NUM_BLOCKS]
  float* d_out;         // [maxBatchSize * NUM_BLOCKS]

  int* d_res_idx;       // [maxBatchSize * maxBeamSize]
  float* d_res;         // [maxBatchSize * maxBeamSize]

  int* h_res_idx;       // [maxBeamSize * maxBatchSize]
  float* h_res;         // [maxBeamSize * maxBatchSize]

  float* d_breakdown;   // [maxBeamSize]
  int* d_batchPosition; // [maxBatchSize + 1]
  int* d_cumBeamSizes;  // [maxBatchSize + 1]
  //size_t lastN;
};

// factory function
// Returns a lambda with the same signature as the getNBestList() function.
GetNBestListFn createGetNBestListGPUFn(size_t beamSize, size_t dimBatch, DeviceId deviceId) {
  auto nth = New<NthElementGPU>(beamSize, dimBatch, deviceId);
  return [nth](Tensor logProbs, size_t N, std::vector<float>& outCosts, std::vector<unsigned>& outKeys, const bool isFirst) {
    return nth->getNBestList(logProbs, N, outCosts, outKeys, isFirst);
  };
}

}  // namespace marian
